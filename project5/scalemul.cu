#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error calling \"" #call "\", code is " << err << std::endl; \
        exit(-1); \
    } \
}
void initializeMatrix(float* matrix, int N) {
    for (int i = 0; i < N * N; i++) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

int main() {
    const int N = 4096;
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;

    A = new float[N * N];
    B = new float[N * N];
    C = new float[N * N];

    initializeMatrix(A, N);
    initializeMatrix(B, N);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc((void **)&d_A, N * N * sizeof(float));
    hipMalloc((void **)&d_B, N * N * sizeof(float));
    hipMalloc((void **)&d_C, N * N * sizeof(float));

    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.0f, beta = 0.0f;
    hipEventRecord(start);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);
    hipEventRecord(stop);

    CUDA_CHECK(hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost));
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "GPU Multiplication Time: " << milliseconds << " ms." << std::endl;

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}
