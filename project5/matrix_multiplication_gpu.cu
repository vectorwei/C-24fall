// matrix_multiplication_gpu.cu
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA error wrapper
#define CUDA_CHECK(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error calling \"" #call "\", code is " << err << std::endl; \
        exit(-1); \
    } \
}

// Initialize matrix with random data
void initializeMatrix(float* matrix, int N) {
    for (int i = 0; i < N * N; i++) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

int main() {
    const int N = 4096;
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;

    A = new float[N * N];
    B = new float[N * N];
    C = new float[N * N];

    initializeMatrix(A, N);
    initializeMatrix(B, N);

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipMalloc((void **)&d_A, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&d_B, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&d_C, N * N * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.0f, beta = 0.0f;
    CUDA_CHECK(hipEventRecord(start));
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);
    CUDA_CHECK(hipEventRecord(stop));

    CUDA_CHECK(hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipEventSynchronize(stop));

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "GPU Matrix Multiplication Time: " << milliseconds << " ms." << std::endl;

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}
